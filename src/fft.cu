#include "fft.h"
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>

// Perform FFT on the GPU and return the result to host
hipfftComplex* perform_fft(hipfftComplex* host_input, int N) {
    hipfftComplex *device_input, *device_output;

    // Allocate memory on the device
    hipMalloc(&device_input, N * sizeof(hipfftComplex));
    hipMalloc(&device_output, N * sizeof(hipfftComplex));

    // Copy data from host to device
    hipMemcpy(device_input, host_input, N * sizeof(hipfftComplex), hipMemcpyHostToDevice);

    // Create plan and execute FFT
    hipfftHandle plan;
    hipfftPlan1d(&plan, N, HIPFFT_C2C, 1);
    hipfftExecC2C(plan, device_input, device_output, HIPFFT_FORWARD);

    // Allocate output memory on the host
    hipfftComplex* host_output = new hipfftComplex[N];

    // Copy result back to host
    hipMemcpy(host_output, device_output, N * sizeof(hipfftComplex), hipMemcpyDeviceToHost);

    // Cleanup
    hipfftDestroy(plan);
    hipFree(device_input);
    hipFree(device_output);

    return host_output;
}

void perform_fft_2d(hipfftComplex* host_input, hipfftComplex* host_output, int width, int height) {
    hipfftComplex *device_input, *device_output;
    hipMalloc(&device_input, width * height * sizeof(hipfftComplex));
    hipMalloc(&device_output, width * height * sizeof(hipfftComplex));
    
    hipMemcpy(device_input, host_input, width * height * sizeof(hipfftComplex), hipMemcpyHostToDevice);

    hipfftHandle plan;
    hipfftPlan1d(&plan, width, HIPFFT_C2C, height); // Plan for batched 1D FFTs
    hipfftExecC2C(plan, device_input, device_output, HIPFFT_FORWARD);

    hipMemcpy(host_output, device_output, width * height * sizeof(hipfftComplex), hipMemcpyDeviceToHost);

    hipfftDestroy(plan);
    hipFree(device_input);
    hipFree(device_output);
}
