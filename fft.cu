#include "hip/hip_runtime.h"
#include "fft.h"
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>

// CUDA kernel to prepare data, assuming input is in row-major format
__global__ void prepareData(hipfftReal* input, hipfftReal* output, int width, int startX, int startY, int windowWidth, int windowHeight) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx < windowWidth && idy < windowHeight) {
        int inputIndex = (startY + idy) * width + (startX + idx);
        output[idy * windowWidth + idx] = input[inputIndex];
    }
}

// Wrapper function to call FFT
void performFFT(hipfftReal* input, hipfftComplex* output, int width, int height, int startX, int startY, int windowWidth, int windowHeight) {
    hipfftHandle plan;
    hipfftReal* d_input;
    hipfftComplex* d_output;

    // Allocate memory on device
    hipMalloc(&d_input, windowWidth * windowHeight * sizeof(hipfftReal));
    hipMalloc(&d_output, windowWidth * windowHeight * sizeof(hipfftComplex));

    // Prepare data
    dim3 blocks((windowWidth + 15) / 16, (windowHeight + 15) / 16);
    dim3 threads(16, 16);
    prepareData << <blocks, threads >> > (input, d_input, width, startX, startY, windowWidth, windowHeight);

    // Create plan and execute FFT
    hipfftPlan2d(&plan, windowWidth, windowHeight, HIPFFT_R2C);
    hipfftExecR2C(plan, d_input, d_output);

    // Copy result back to host
    hipMemcpy(output, d_output, windowWidth * windowHeight * sizeof(hipfftComplex), hipMemcpyDeviceToHost);

    // Cleanup
    hipfftDestroy(plan);
    hipFree(d_input);
    hipFree(d_output);
}
